
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define BLOCK_SIZE 32

const int DSIZE = 256;
const int a = 1;
const int b = 1;

// error checking macro
#define cudaCheckErrors()                                       \
	do {                                                        \
		hipError_t __err = hipGetLastError();                 \
		if (__err != hipSuccess) {                             \
			fprintf(stderr, "Error:  %s at %s:%d \n",           \
			hipGetErrorString(__err),__FILE__, __LINE__);      \
			fprintf(stderr, "*** FAILED - ABORTING***\n");      \
			exit(1);                                            \
		}                                                       \
	} while (0)


// CUDA kernel that runs on the GPU
__global__ void dot_product(const int *A, const int *B, int *C, int N) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x; 
	// FIXME
	// Use atomicAdd
    if (idx < N){
        int tmp = A[idx] * B[idx];
        atomicAdd(C, tmp);
    }
}


int main() {
	
	// Create the device and host pointers
	int *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;

	// Fill in the host pointers 
	h_A = new int[DSIZE];
	h_B = new int[DSIZE];
	h_C = new int;
	for (int i = 0; i < DSIZE; i++){
		h_A[i] = a;
		h_B[i] = b;
	}

	*h_C = 0;


	// Allocate device memory
    hipMalloc(&d_A, DSIZE * sizeof(int));
    hipMalloc(&d_B, DSIZE * sizeof(int));
    hipMalloc(&d_C, sizeof(int));
	
	// Check memory allocation for errors
    cudaCheckErrors();
        
	// Copy the matrices on GPU
    hipMemcpy(d_A, h_A, DSIZE*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, DSIZE*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, sizeof(int), hipMemcpyHostToDevice);
	
	// Check memory copy for errors
    cudaCheckErrors();
    
	// Define block/grid dimentions and launch kernel
    int block_size = 256;
    int grid_size = (DSIZE + block_size - 1)/block_size;
    dot_product<<<grid_size, block_size>>>(d_A, d_B, d_C, DSIZE);
    hipDeviceSynchronize();
    cudaCheckErrors();
    
	// Copy results back to host
	hipMemcpy(h_A, d_A, DSIZE*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_B, d_B, DSIZE*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_C, d_C, sizeof(int), hipMemcpyDeviceToHost);
    
    // Check copy for errors
    cudaCheckErrors();
    
	// Verify result
    printf("C is calculated to be %d \n", *h_C);
    
	// Free allocated memory
    free(h_A);
    free(h_B);
    free(h_C);     

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
	
	return 0;

}